#include "hip/hip_runtime.h"
﻿
#include "CircleDetectorCuda.h"

//#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


void checkCudaError(hipError_t err, const char* msg) 
{

    if (err != hipSuccess) 
    {
        std::cerr << "Error: " << msg << " (" << hipGetErrorString(err) << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void DetectCircleAtKernel(const unsigned char* img, const unsigned char* mask, uint16_t* response, int w, int h, int mw, int mh, int diff_threshold, int count_threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    int mw2 = mw / 2;
    int mh2 = mh / 2;

    int c = img[y * w + x];
    int count = 0;
    uint16_t resp = 0;

    for (int64_t yy = 0; yy < mh; ++yy) 
    {
        for (int64_t xx = 0; xx < mw; ++xx) 
        {
            if (mask[yy * mw + xx]) 
            {
                int16_t j = y + yy - mw2;
                int16_t i = x + xx - mh2;

                if (i>-1 && j>-1 && i < w && j < h)
                {
                    int v = img[j * w + i];

                    int d = abs(c - v);
                    //int d = (v - c);
                    if ( d >= diff_threshold)
                    {
                        count += 1;
                        resp += d;
                    }
                }
            }
        }
    }

    if (count >= count_threshold) 
        response[y * w + x] = resp;    
    else
        response[y * w + x] = 0;
}

CircleDetectorCuda::CircleDetectorCuda(const DetectionParams& params, int img_width, int img_height): CircleDetector(params, img_width, img_height)
{    
    checkCudaError(hipMalloc((void**)&img_dev,  W * H * sizeof(unsigned char)), "hipMalloc img_dev");
    checkCudaError(hipMalloc((void**)&mask_dev, MW * MH * sizeof(unsigned char)), "hipMalloc mask_dev");
    checkCudaError(hipMalloc((void**)&resp_dev, W * H * sizeof(uint16_t)), "hipMalloc resp_dev");

    checkCudaError(hipMemcpy(mask_dev, Mask.data, MW * MH * sizeof(unsigned char), hipMemcpyHostToDevice), "hipMemcpy mask_dew");
}


CircleDetectorCuda::~CircleDetectorCuda()
{
    hipFree(img_dev);
    hipFree(mask_dev);
    hipFree(resp_dev);
}

void CircleDetectorCuda::CalcResponse(const cv::Mat& image)
{
    ImagePtr = image.data;

    // copy data
    checkCudaError(hipMemcpy(img_dev, image.data, W * H * sizeof(unsigned char), hipMemcpyHostToDevice), "hipMemcpy dev_input");

    dim3 blockDims(16, 16);
    dim3 gridDims((W + blockDims.x - 1) / blockDims.x, (H + blockDims.y - 1) / blockDims.y);

    // execute detection
    DetectCircleAtKernel<<< gridDims, blockDims >>>(img_dev, mask_dev, resp_dev, W, H, MW, MH, Params.DiffThreshold, Params.CountThreshold);

    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution");

    //copy to host
    checkCudaError(hipMemcpy(CircleResponse.data, resp_dev, W * H * sizeof(uint16_t), hipMemcpyDeviceToHost), "hipMemcpy dev_output");
}